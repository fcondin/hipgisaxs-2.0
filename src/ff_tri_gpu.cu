#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>


#include "types.h"
#include "cuda_utils.cuh"
#include "cmplx_utils.cuh"
#include "rotation.h"

namespace hig {


    __device__ cucomplex_t 
    ffTriangle (const triangle_t & tri, const cucomplex_t * q){

        // constants
        constexpr double TINY = 1.0E-20;
        const cucomplex_t j1 = make_cucomplex(0, 1);
        const cucomplex_t jn = make_cucomplex(0,-1);

        // initialize stuff
        cucomplex_t ff = 0;

        // calculate q^2
        double q_sq = norm3(q);

        // form vertices
        vector3_t vertex[3];
        vertex[0] = vector3_t(tri.v1.x_, tri.v1.y_, tri.v1.z_);
        vertex[1] = vector3_t(tri.v2.x_, tri.v2.y_, tri.v2.z_);
        vertex[2] = vector3_t(tri.v3.x_, tri.v3.y_, tri.v3.z_);

        // form edges
        vector3_t edge[3];
        edge[0] = vertex[1] - vertex[0];
        edge[1] = vertex[2] - vertex[1];
        edge[2] = vertex[0] - vertex[2];

        // calculate outward normal and normalize
        vector3_t n_t = cross(edge[0], edge[1]);
        double  t_area = 0.5 * n_t.abs();
        n_t /= 2 * t_area;
 
        // dot (q, n_t)
        cucomplex_t q_dot_nt = dot(q, n_t);

        // calculate projection
        double proj_tq = q_sq - cuda::std::norm(q_dot_nt);

        // CASE 1
        if (cuda::std::abs(proj_tq) < TINY){
            cucomplex_t q_dot_v = dot(q, vertex[0]);

            // calculate Form-Factor
            ff = j1 * q_dot_nt * t_area / q_sq * cuda::std::exp(jn * q_dot_v);
        } else {

            // iterate on each edge :
            for (int e = 0; e < 3; e++) {

                // edge normal
                vector3_t n_e = cross(edge[e], n_t);
                n_e /= n_e.abs(); // normalize

                // dot(q, n_e)
                cucomplex_t q_dot_ne = dot(q, n_e);

                // proj_ne
                double proj_eq = proj_tq - cuda::std::norm(q_dot_ne);

                // CASE 2
                if (cuda::std::abs(proj_eq) < TINY){

                    // q_dot_v
                    cucomplex_t q_dot_v = dot(q, vertex[e]);

                    // calculate contribution of edge
                    double f0 = edge[e].abs() / (q_sq * proj_tq);
                    cucomplex_t f1 = - q_dot_nt * q_dot_ne;
                    cucomplex_t f2 = cuda::std::exp(jn * q_dot_v);
                    auto tmp = (f0 * f1 * f2);
                    ff += tmp;
                } else {
                    // CASE 3 (General case)
                    int e1 = (e+1) % 3;
                    double f0 = q_sq * proj_tq * proj_eq;
                    cucomplex_t f1 = jn * q_dot_nt * q_dot_ne;
                    cucomplex_t f2 = dot(q, edge[e]) / edge[e].abs();
                    cucomplex_t t1 = dot(q, vertex[e]);
                    cucomplex_t f3 = cuda::std::exp(jn * t1);
                    cucomplex_t t2 = dot(q, vertex[e1]);
                    cucomplex_t f4 = cuda::std::exp(jn * t2);
                    auto tmp = f1 * f2 * (f3-f4) / f0;
                    ff += tmp;
                }
            }
        }
        return ff;
    }

    __global__ void ff_tri_kernel1 (
            unsigned int nq, 
            const double *qx, const double *qy, const cucomplex_t *qz,
            int num_triangles, const triangle_t * triangles,
            const RotMatrix_t rot, cucomplex_t * ff) {

        unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
        if ( i < nq ) {
            cucomplex_t mq[3];
            rot.rotate(qx[i], qy[i], qz[i], mq);
            ff[i] = 0;
            for (int j=0; j < num_triangles; j++)
                ff[i] += ffTriangle(triangles[j], mq);
        } 
    } 

  /**
   * The main host function called from outside, as part of the API for a single node.
   */
  void FFTriangulation(
            int num_triangles, const triangle_t * triangles,
            int nq, const double * qx, const double * qy, 
            const cucomplex_t * qz, const RotMatrix_t rot,
            cucomplex_t *ff) { 
      
        double *d_qx, *d_qy;
        cucomplex_t *d_qz;
        cucomplex_t *d_ff;
        triangle_t *d_triangles;

        // allocate space for ff
        SAFE_CALL(hipMalloc((void **) &d_ff, nq * sizeof(cucomplex_t)));

        // Allocate memory for qx, qy, qz
        SAFE_CALL(hipMalloc((void **) &d_qx, nq * sizeof(cucomplex_t)));
        SAFE_CALL(hipMalloc((void **) &d_qy, nq * sizeof(cucomplex_t)));
        SAFE_CALL(hipMalloc((void **) &d_qz, nq * sizeof(cucomplex_t)));
        // Allocate memory for triangles
        SAFE_CALL(hipMalloc((void **) &d_triangles, num_triangles * sizeof(triangle_t)));

        // copy buffers to device memory
        SAFE_CALL(hipMemcpy(d_qx, qx, nq * sizeof(double), hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(d_qy, qy, nq * sizeof(double), hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(d_qz, qz, nq * sizeof(cucomplex_t), hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(d_triangles, triangles, num_triangles * sizeof(triangle_t), hipMemcpyHostToDevice));



        // number of cuda threads
        int num_threads = 256;
        int num_blocks = nq % num_threads ? nq/num_threads+1 : nq/num_threads;

        // Kernel 1
        ff_tri_kernel1 <<< num_blocks, num_threads >>> (
                nq, d_qx, d_qy, d_qz, 
                num_triangles, d_triangles, rot, d_ff);
        SAFE_CALL(hipGetLastError());

        // allocate memory to computed ff
        SAFE_CALL(hipMemcpy(ff, d_ff, nq * sizeof(cucomplex_t), hipMemcpyDeviceToHost));
 
        hipFree(d_triangles);
        hipFree(d_qz);
        hipFree(d_qy);
        hipFree(d_qx);
        hipFree(d_ff);
    } 
}

